//
// Created by liuzhenhai on 28.2.23.
//



#include <hip/hip_runtime.h>
#include <stdio.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

// grid stride loop
__global__ void AddKernel(float *x, float *y, float *z, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        z[i] = x[i] + y[i];
    }
}

void Add1d(float *x, float *y, float *z, int n) {
    for (int i = 0; i < n; i++) {
        z[i] = x[i] + y[i];
    }
}

void Add2d(float *x, float *y, float *z, int m, int n) {
    Add1d(x, y, z, m * n);
}


void Add1dWithKernel(float *x, float *y, float *z, int n) {
    float *x_d, *y_d, *z_d;
    // allocate gpu memory
    HANDLE_ERROR(hipMalloc( (void**)&x_d, n * sizeof(float) ));
    HANDLE_ERROR(hipMalloc( (void**)&y_d, n * sizeof(float) ));
    HANDLE_ERROR(hipMalloc( (void**)&z_d, n * sizeof(float) ));
    HANDLE_ERROR(hipMemcpy(x_d, x, n * sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(y_d, y, n * sizeof(float), hipMemcpyHostToDevice));
    constexpr int block_size = 256;
    constexpr int grid_size = 1024;
    AddKernel<<<grid_size, block_size>>>(x_d, y_d, z_d, n);
    hipMemcpy(y, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
    // free the memory allocated on the GPU
    HANDLE_ERROR(hipFree( x_d ));
    HANDLE_ERROR(hipFree( y_d ));
    HANDLE_ERROR(hipFree( z_d ));
}

void Add2dWithKernel(float *x, float *y, float *z, int m, int n) {
    Add1dWithKernel(x, y, z, m *  n);
}
