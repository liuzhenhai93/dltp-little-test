
#include <hip/hip_runtime.h>
//
// Created by liuzhenhai on 28.2.23.
//

// grid stride loop
__global__ void AddKernel(float *x, float *y, float *z, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        z[i] = x[i] + y[i];
    }
}

void Add1d(float *x, float *y, float *z, int n) {
    for (int i = 0; i < n; i++) {
        z[i] = x[i] + y[i];
    }
}

void Add2d(float *x, float *y, float *z, int m, int n) {
    Add1d(x, y, z, m * n);
}


void Add1dWithKernel(float *x, float *y, float *z, int n) {
    float *x_d, *y_d, *z_d;
    // allocate gpu memory
    hipMalloc( (void**)&x_d, n * sizeof(float) );
    hipMalloc( (void**)&y_d, n * sizeof(float) );
    hipMalloc( (void**)&z_d, n * sizeof(float) );
    hipMemcpy(x_d, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, n * sizeof(float), hipMemcpyHostToDevice);
    constexpr int block_size = 256;
    constexpr int grid_size = 1024;
    AddKernel<<<grid_size, block_size>>>(x_d, y_d, z_d, n);
    hipMemcpy(y, y_d, n * sizeof(float), hipMemcpyDeviceToHost);
    // free the memory allocated on the GPU
    hipFree( x_d );
    hipFree( y_d );
    hipFree( z_d );
}

void Add2dWithKernel(float *x, float *y, float *z, int m, int n) {
    Add1dWithKernel(x, y, z, m *  n);
}